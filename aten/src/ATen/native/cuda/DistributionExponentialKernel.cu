#include <ATen/AccumulateType.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <functional>
#include <utility>

#include <ATen/LegacyTHFunctionsCUDA.h>
#include <ATen/native/Distributions.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>

#include <THC/THCGeneral.h>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <limits>
#include <type_traits>
#include <utility>

namespace at {
namespace native {

void exponential_kernel(
    TensorIteratorBase& iter,
    double lambda,
    c10::optional<Generator> gen) {
  auto generator = get_generator_or_default<CUDAGeneratorImpl>(
      gen, cuda::detail::getDefaultCUDAGenerator());
  at::native::templates::cuda::exponential_kernel(iter, lambda, generator);
}

REGISTER_DISPATCH(exponential_stub, &exponential_kernel);

} // namespace native
} // namespace at
